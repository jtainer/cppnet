// 
// Device layer member function definitions
// 
// 2022, Jonathan Tainer
//

#include "network.h"
#include <hip/hip_runtime.h>

DevLayer::DevLayer() {
	weightMatrix = NULL;
	outputVector = NULL;
	delta = NULL;
	numOfNodes = 0;
	weightsPerNode = 0;
}

DevLayer::DevLayer(const DevLayer& source) {
	numOfNodes = source.numOfNodes;
	weightsPerNode = source.weightsPerNode;

	hipMalloc((void**)&weightMatrix, sizeof(float) * numOfNodes * weightsPerNode);
	hipMalloc((void**)&outputVector, sizeof(float) * numOfNodes);
	hipMalloc((void**)&delta, sizeof(float) * numOfNodes);

	hipMemcpy(weightMatrix, source.weightMatrix, sizeof(float) * numOfNodes * weightsPerNode, hipMemcpyDeviceToDevice);
	hipMemcpy(outputVector, source.outputVector, sizeof(float) * numOfNodes, hipMemcpyDeviceToDevice);
	hipMemcpy(delta, source.delta, sizeof(float) * numOfNodes, hipMemcpyDeviceToDevice);
}

DevLayer::DevLayer(const Layer& source) {
	numOfNodes = source.numOfNodes;
	weightsPerNode = source.weightsPerNode;

	hipMalloc((void**)&weightMatrix, sizeof(float) * numOfNodes * weightsPerNode);
	hipMalloc((void**)&outputVector, sizeof(float) * numOfNodes);
	hipMalloc((void**)&delta, sizeof(float) * numOfNodes);

	hipMemcpy(weightMatrix, source.weightMatrix, sizeof(float) * numOfNodes * weightsPerNode, hipMemcpyHostToDevice);
	hipMemcpy(outputVector, source.outputVector, sizeof(float) * numOfNodes, hipMemcpyHostToDevice);
	hipMemcpy(delta, source.delta, sizeof(float) * numOfNodes, hipMemcpyHostToDevice);
}

DevLayer::DevLayer(const int nodes, const int inputs) {
	numOfNodes = nodes;
	weightsPerNode = inputs;

	hipMalloc((void**)&weightMatrix, sizeof(float) * numOfNodes * weightsPerNode);
	hipMalloc((void**)&outputVector, sizeof(float) * numOfNodes);
	hipMalloc((void**)&delta, sizeof(float) * numOfNodes);
}

DevLayer& DevLayer::operator=(const DevLayer& source) {
	if (this == &source) {
		return *this;
	}
	
	// Reallocate memory only if needed
	if (numOfNodes != source.numOfNodes || weightsPerNode != source.weightsPerNode) {
		hipFree(weightMatrix);
		hipFree(outputVector);
		hipFree(delta);
	
		numOfNodes = source.numOfNodes;
		weightsPerNode = source.weightsPerNode;
	
		hipMalloc((void**)&weightMatrix, sizeof(float) * numOfNodes * weightsPerNode);
		hipMalloc((void**)&outputVector, sizeof(float) * numOfNodes);
		hipMalloc((void**)&delta, sizeof(float) * numOfNodes);
	}
	
	hipMemcpy(weightMatrix, source.weightMatrix, sizeof(float) * numOfNodes * weightsPerNode, hipMemcpyDeviceToDevice);
	hipMemcpy(outputVector, source.outputVector, sizeof(float) * numOfNodes, hipMemcpyDeviceToDevice);
	hipMemcpy(delta, source.delta, sizeof(float) * numOfNodes, hipMemcpyDeviceToDevice);
	
	return *this;
}

DevLayer& DevLayer::operator=(const Layer& source) {
	
	// Reallocate memory only if needed
	if (numOfNodes != source.numOfNodes || weightsPerNode != source.weightsPerNode) {
		hipFree(weightMatrix);
		hipFree(outputVector);
		hipFree(delta);
	
		numOfNodes = source.numOfNodes;
		weightsPerNode = source.weightsPerNode;
	
		hipMalloc((void**)&weightMatrix, sizeof(float) * numOfNodes * weightsPerNode);
		hipMalloc((void**)&outputVector, sizeof(float) * numOfNodes);
		hipMalloc((void**)&delta, sizeof(float) * numOfNodes);
	}
	
	hipMemcpy(weightMatrix, source.weightMatrix, sizeof(float) * numOfNodes * weightsPerNode, hipMemcpyHostToDevice);
	hipMemcpy(outputVector, source.outputVector, sizeof(float) * numOfNodes, hipMemcpyHostToDevice);
	hipMemcpy(delta, source.delta, sizeof(float) * numOfNodes, hipMemcpyHostToDevice);
	
	return *this;
}

DevLayer::~DevLayer() {
	hipFree(weightMatrix);
	hipFree(outputVector);
	hipFree(delta);
}
